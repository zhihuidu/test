/**
 * @brief Clustering coefficient test program
 * @file
 */

#include "HornetAlg.hpp"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>

#include "Static/ClusteringCoefficient/cc.cuh"

using namespace timer;
using namespace hornets_nest;

using HornetGraph = ::hornet::gpu::Hornet<vid_t>;

int exec(int argc, char* argv[]) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);
    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);
    //hornet_graph.print();
    ClusteringCoefficient cc(hornet_graph);
    cc.init();

    Timer<DEVICE> TM(5);
    TM.start();

    cc.run();

    TM.stop();
    TM.print("Computation time:");
  
    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
    hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.

    ret = exec(argc, argv);

    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    hornets_nest::gpu::finalizeRMMPoolAllocation();

    return ret;
}

